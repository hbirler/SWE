#include "MemoryManager.hh"

#include <hip/hip_runtime.h>

static void checkCUDAErrorX(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "\nCuda error (%s): %s.\n", msg, hipGetErrorString( err) );
    exit(-1);
  }
}

void* swe_alloc(size_t alignment, size_t elem, size_t count) {
  void *ptr;
  hipMallocManaged(&ptr, elem * count, hipMemAttachGlobal);
  checkCUDAErrorX("allocate managed memory");
  return ptr;
}

void swe_free(void* ptr) {
  hipFree(ptr);
  checkCUDAErrorX("free managed memory");
}